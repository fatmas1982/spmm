#include "hip/hip_runtime.h"
/**
 * @author : Sean Rose (http://www.cs.fsu.edu/research/projects/rose_report.pdf)
 * 
 * Implementation of SPMM CUDA Kernels
 *
 */

__global__ void GetNNZ(sparse_matrix A, sparse_matrix B, sparse_matrix C, int* workingSet)
{
	const int laneId = threadIdx.x;
	const int warpId = blockIdx.x;
	
	int* nonzeros;
	int rowAStart, rowAEnd, rowBStart, rowBEnd;
	int nnz;
	int colC;
	
	extern __shared__ int nzCount[];
	
	nonzeros = &workingSet[warpId * B.cols];
	
	// Iterate through each assigned row in A.
	for(int rowA = warpId; rowA < A.rows; rowA += gridDim.x)
	{
		rowAStart = A.ir[rowA];
		rowAEnd = A.ir[rowA + 1];
		// There are no non-zeros in this row so continue
		if(rowAStart == rowAEnd)
		{
			if (laneId == 0)
				C.ir[rowA] = 0;
			__syncthreads();
			continue;
		}

		// Reset the nz counts
		nzCount[laneId] = 0;
		
		// reset the nonzeros table
		for (int i=laneId; i<B.cols; i+= warpSize)
		{
			nonzeros[i] = 0;
		}
		__syncthreads();
		
		for(int i = rowAStart; i < rowAEnd; ++i)
		{
			rowBStart = B.ir[A.jc[i]];
			rowBEnd = B.ir[A.jc[i]+1];

			for (int j = rowBStart + laneId; j < rowBEnd; j += warpSize)
			{
				colC = B.jc[j];
				nzCount[laneId] += nonzeros[colC] == 0;
				nonzeros[colC] = 1;
			}
			__syncthreads();
		}

		if(laneId == 0)
		{
			nnz = nzCount[0];
			for(int i = 1; i < warpSize; ++i)
			{
				nnz += nzCount[i];
			}
			C.ir[rowA] = nnz;

		}
		
		__syncthreads();
	}
}

__global__ void GetVals(sparse_matrix A, sparse_matrix B, sparse_matrix C, int* indexTable)
{
	const int laneId = threadIdx.x;
	const int bloackId = blockIdx.x;
	
	__shared__ unsigned int back;
	
	int rowAStart; // The index into A.jc and A.val
	int rowAEnd; // The boundary index for A
	float valA; // The value of the current A nonzero
	int rowBStart; // The index into B.jc and B.val
	int rowBEnd; // The boundary index for B
	int colB; // The current column in B being used
	int rowCStart; // The index into C.jc and C.val
	int rowCEnd; // The boundary index for C
	int hash; // The calculated hash value
	int i, j; // Loop iterators

	// Set the global hash table to point to the space
	// used by this warp
	int* gColHashTable;
	float* gValHashTable;
	int globalEntries;
	
	indexTable = &indexTable[C.cols * blockId];
	
	if(laneId == 0)
		back = 0;
	
	for(int rowA = blockId; rowA < A.rows; rowA += gridDim.x)
	{
		rowAStart = A.ir[rowA];
		rowAEnd = A.ir[rowA + 1];
		for(i = laneId; i < C.cols; ++i)
		{
			indexTable[i] = -1;
		}
		__syncthreads();

		// Set the location of the global hash table
		rowCStart = C.ir[rowA];
		rowCEnd = C.ir[rowA + 1];
		globalEntries = rowCEnd - rowCStart;
		gColHashTable = &C.jc[rowCStart];
		gValHashTable = &C.val[rowCStart];
		for(i = rowAStart; i < rowAEnd; ++i)
		{
			valA = A.val[i];
			rowBStart = B.ir[A.jc[i]];
			rowBEnd = B.ir[A.jc[i] + 1];
			int curIdx;
			int* storeInt;
			float* storeFloat;
			float valB;
			for(j = rowBStart + laneId; __any(j < rowBEnd); j += warpSize)
			{
				colB = j < rowBEnd ? B.jc[j] : -1;
				curIdx = colB == -1 ? -1 : indexTable[colB];
				hash = colB != -1 && curIdx == -1 ? atomicInc(&back, globalEntries - 1) : curIdx;
				storeInt = hash == -1 ? &hash : &indexTable[colB];
				*storeInt = hash;
				storeInt = hash == -1 ? &colB : &gColHashTable[hash];
				*storeInt = colB;
				valB = colB == -1 ? 1 : B.val[j];
				storeFloat = hash == -1 ? &valA : &gValHashTable[hash];
				*storeFloat += valB * valA;
			}
		} // For each nonzero in the A row
	} // For each assigned row in A
}

__global__ void SortCols(sparse_matrix C, int maxRowNNZ, int* workQueue)
{
	const int laneId = threadIdx.x;
	const int blockId = blockIdx.x;
	
	// Dynamic shared memory
	extern __shared__ int sharedMem[];
	
	// The maximum size of the queue
	const int queueSize = (maxRowNNZ / 2) + 1;
	
	// The maximum number of passes needed
	int maxShift = __log2f(C.cols) / RADIX_BITS;
	
	// The number of passes for the work in the queue
	int* workPasses = &workQueue[blockId * queueSize];
	
	// The front of the bucket for the work in the queue
	int* workFronts = &workQueue[gridDim.x * queueSize];
	workFronts = &workFronts[blockId * queueSize];
	
	// The back of the bucket for the work in the queue
	int* workBacks = &workQueue[gridDim.x * queueSize * 2];
	workBacks = &workBacks[blockId * queueSize];
	int front; // The front of the work queue.
	__shared__ unsigned int back; // The back of the work queue.

	// Holds the sizes for the buckets being sorted by the threads
	int* bucketSizes = &sharedMem[laneId * RADIX_BASE];

	// The ending index of the buckets being sorted
	int* bucketBounds = &sharedMem[blockDim.x * RADIX_BASE];
	bucketBounds = &bucketBounds[laneId * RADIX_BASE];
	int pass; // The pass number of the current bucket
	int bucketFront; // The index of the front of the bucket
	int bucketBack; // The index of the back of the bucket
	int bucketIdx; // The index of an item in the bucket
	int shiftCount; // The number of bits to shift to get the index
	int iTmp; // A temporary variable for swapping
	float fTmp;
	int swapIdx; // The index to swap with
	int queueIdx; // An index into the work queue
	int prev; // The previous bucket offset
	int subIdx;

	for(int rowC = blockId; rowC < C.rows; rowC += gridDim.x)
	{
		// Skip if there are not non-zeros to sort
		if(C.ir[rowC] == C.ir[rowC + 1])
			continue;
		
		// Clear the work queue
		for(int i = laneId + 1; i < queueSize; i += blockDim.x)
		{
			workPasses[i] = -1;
		}
		workPasses[0] = 0;
		workFronts[0] = C.ir[rowC];
		workBacks[0] = C.ir[rowC + 1];
		front = 0;
		back = 1;
		__syncthreads();
		
		// While there is more work in the queue
		while(front != back)
		{
			queueIdx = (front + laneId) % queueSize;
			
			// Get the work
			pass = workPasses[queueIdx];
			bucketFront = workFronts[queueIdx];
			bucketBack = workBacks[queueIdx];
			
			// Clear this work
			workPasses[queueIdx] = -1;
			
			// Move the front forward
			if((back > front && back - front <= blockDim.x) || (back < front && (back + queueSize) - front <= blockDim.x))
			{
				front = back;
			}
			else
			{
				front = (front + blockDim.x) % queueSize;
			}
			
			// There is work to do
			if(pass >= 0)
			{
				// Clear the bucket sizes
				for(int i = 0; i < RADIX_BASE; ++i)
				{
					bucketSizes[i] = 0;
				}
				shiftCount = (maxShift - pass) * RADIX_BITS;
				
				// First, determine the size of the buckets
				for(int i = bucketFront; i < bucketBack; ++i)
				{
					++bucketSizes[(C.jc[i] >> shiftCount) & RADIX_MASK];
				}
				
				// Determine the indexes of the buckets and put
				// them into the work queue
				prev = bucketFront;
				for(int i = 0; i < RADIX_BASE; ++i)
				{
					// Determine the bucket end
					bucketIdx = bucketSizes[i] + prev;

					// Place the bucket into the work queue only
					// if it has items to be sorted
					if(bucketSizes[i] > 1)
					{
						queueIdx = atomicInc(&back, queueSize - 1);
						workPasses[queueIdx] = pass + 1;
						workFronts[queueIdx] = prev;
						workBacks[queueIdx] = bucketIdx;
					}
					
					// Store the bucket end
					bucketSizes[i] = bucketIdx;
					bucketBounds[i] = bucketIdx;
					prev = bucketIdx;
				}

				// Place the items into the buckets
				bucketIdx = bucketFront;
				while(bucketIdx != bucketBack)
				{
					subIdx = (C.jc[bucketIdx] >> shiftCount) & RADIX_MASK;
					swapIdx = --bucketSizes[subIdx];;
					
					// Done sorting this bucket, move to the next open one
					if(swapIdx == bucketIdx)
					{
						do 
						{
							bucketIdx = bucketBounds[subIdx++];
						} while(bucketIdx != bucketBack && bucketSizes[subIdx] == bucketIdx);
					}
					else
					{
						// Swap swapIdx and bucketIdx
						iTmp = C.jc[swapIdx];
						C.jc[swapIdx] = C.jc[bucketIdx];
						C.jc[bucketIdx] = iTmp;
						fTmp = C.val[swapIdx];
						C.val[swapIdx] = C.val[bucketIdx];
						C.val[bucketIdx] = fTmp;
					}
				}
			} // If this thread has work
			__syncthreads();
		} // While there is work to do
	} // For all rows in C
}